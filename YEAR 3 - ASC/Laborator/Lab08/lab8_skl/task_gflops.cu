#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

__global__ void kernel_gflops(float* a, float* b, int N) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	int idx = row * N + col;

	// ~TODO~
	// Execute at least two floating point operations (e.g. +, -, *, /)
	// on the value from array a at index idx and
	// store the result in array b at index idx. 
	// NOTE: Do not change value directly in array a.
	for (int i = 0; i < 500; i++) {
		b[idx] = (a[idx] + 10.f) * (a[idx] / 2.f) - 5.f;
	}
}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);
    hipSetDevice(0);

    float *device_a = 0;
    float *device_b = 0;
    float *host_a = 0;
    float *host_b = 0;

    int N = 4096;
    int size = N * N;

    // Arrays a and b are of size N * N
    host_a = (float *) malloc(size * sizeof(float));
    host_b = (float *) malloc(size * sizeof(float));
    hipMalloc((void **) &device_a, size * sizeof(float));
    hipMalloc((void **) &device_b, size * sizeof(float));

    if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
        printf("[HOST] Couldn't allocate memory\n");
    	return 1;
    }

    // Populate array a randomly
    fill_array_random(host_a, size);
    hipMemcpy(device_a, host_a, size * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;

    // ~TODO~
    // Create two cuda events (start and stop)
    // by using the hipEventCreate function.
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel_gflops<<<size / 256, 256>>> (device_a, device_b, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    float seconds = ms / pow((float) 10, 3);

    // ~TODO~
    // Set num_ops to the number of floating point operations
    // done in the kernel multiplied with the size of the matrix.
    long num_ops = (long)(1500 * (long)size);

    float gflops = (float) num_ops / seconds / 1e+9;
    printf("GFLOPS: %.3f\n", gflops);

    free(host_a);
    free(host_b);
    hipFree(device_a);
    hipFree(device_b);
    
    return 0;
}
