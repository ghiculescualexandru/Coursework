
#include <hip/hip_runtime.h>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>

#define NUM_ELEM    128

using namespace std;

// TODO
// workers will compute sum on first N elements
__global__ void worker(int *res, int*data, int idx)
{
    // TODO, compute sum and store in result
    atomicAdd(&res[idx], data[threadIdx.x]);
}

// TODO
// master will launch threads to compute sum on first N elements
__global__ void master(int *res, int *data)
{
    // TODO, schedule worker threads
    worker<<<1, data[threadIdx.x]>>>(res, data, threadIdx.x);   	
}

void generateData(int *data, int num) {
    srand(time(0));
    
    for(int i = 0; i < num; i++) {
        data[i] = rand() % 8 + 2;
    }
}

void print(int *data, int num) {
    for(int i = 0; i < num; i++) {
        cout << data[i] << " ";
    }
    cout << endl;
}

// TASK check
// each element result[i] should be sum of first data[i] elements of data[i]
bool checkResult(int *data, int num, int *result) {

    for(int i = 0; i < num; i++) {
        
        int sum = 0;
        for(int j = 0; j < data[i]; j++) {
            sum += data[j];
        }
        
        if(result[i] != sum) {
            cout << "Error at " << i << ", requested sum of first " 
                << data[i] << " elem, got " << result[i] << endl;
            return false;
        }
    }
    
    return true;
}

int main(int argc, char *argv[])
{
    int *data = NULL;
    hipMallocManaged(&data, NUM_ELEM * sizeof(int));

    int *result = NULL;
    hipMallocManaged(&result, NUM_ELEM * sizeof(int));
    
    generateData(data, NUM_ELEM);
    
    // TODO schedule master threads and pass data/result/num 
    master<<< 1, NUM_ELEM >>>(result, data);
    hipDeviceSynchronize();
    
    print(data, NUM_ELEM);
    print(result, NUM_ELEM);
    
    if(checkResult(data, NUM_ELEM, result)) {
        cout << "Result OK" << endl;
    } else { 
        cout << "Result ERR" << endl;
    }
    
    hipFree(data);
    hipFree(result);

    return 0;
}
