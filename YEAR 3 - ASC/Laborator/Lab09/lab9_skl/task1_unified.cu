#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

#define NUM_ELEM    (16 * 1024 * 1024)

__global__ void kernel_compute(float* a, float* b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_UnifiedMem(void) {
    float *host_a = 0;
    float *host_b = 0;

    // TODO-unified: Alocare memorie unificata
    hipMallocManaged(&host_a, NUM_ELEM * sizeof(*host_a));
    hipMallocManaged(&host_b, NUM_ELEM * sizeof(*host_b));

    fill_array_random(host_b, NUM_ELEM);

    kernel_compute<<<NUM_ELEM / 256, 256>>> (host_a, host_b, NUM_ELEM);
    
    // TODO-unified: Dealocare memorie unificata
    hipFree(host_a);
    hipFree(host_b);    

    return 0;
}

int compute_NoUnifiedMem(void) {
    float *device_a = 0;
    float *device_b = 0;
    float *host_a = 0;
    float *host_b = 0;

    host_a = (float *) malloc(NUM_ELEM * sizeof(float));
    host_b = (float *) malloc(NUM_ELEM * sizeof(float));
    
    // TODO-plain: Alocare memorie (GPU/VRAM)
   hipMalloc(&device_a, NUM_ELEM * sizeof(*device_a));
   hipMalloc(&device_b, NUM_ELEM * sizeof(*device_b)); 

   if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
        printf("[HOST] Couldn't allocate memory\n");
    	return 1;
    }

    fill_array_random(host_b, NUM_ELEM);
    
    // TODO-plain: Copiere date host_b (CPU/RAM) => device_b (GPU/VRAM)
    
    hipMemcpy(device_b, host_b, NUM_ELEM, hipMemcpyHostToDevice);

    kernel_compute<<<NUM_ELEM / 256, 256>>> (device_a, device_b, NUM_ELEM);
    
    // TODO-plain: Copiere device_a (GPU/VRAM) => date host_a (CPU/RAM)
    
    hipMemcpy(host_a, device_a, NUM_ELEM, hipMemcpyDeviceToHost);

    free(host_a);
    free(host_b);

    // TODO-plain: Dealocare memorie (GPU/VRAM)
    hipFree(device_a);
    hipFree(device_b);
 
    return 0;
}

int main(void) {
#ifdef NO_UNIFIED_MEMORY
    compute_NoUnifiedMem();
#else
    compute_UnifiedMem();
#endif
}
