
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define INDEX_NUM   3

#define INDEX_SUM   0
#define INDEX_MAX   1
#define INDEX_MIN   2

#define NUM_MAX         1024

#define ITEMS_NUM       (1024 * 1024)
#define BLOCK_SIZE      256

using namespace std;

// TODO-1 => ./task_no_atomic
// 1 thread does all compute, no atomic/sync
// thread.0 of block.0 computes everything
__global__ void kernel_no_atomics(int *data, int *results) {
	for (int i = 0; i < ITEMS_NUM; i++) {
		results[INDEX_SUM] += data[i];
	
		if (data[i] > results[INDEX_MAX]) results[INDEX_MAX] = data[i];
		if (data[i] < results[INDEX_MIN]) results[INDEX_MIN] = data[i];
	}
}

// TODO-2 => ./task_partial_atomic
// ITEMS_NUM / 256 threads, ITEMS_NUM / 256 * 3 atomic calls
// thread.0 of each block does partial compute, than uses atomics to compute
__global__ void kernel_partial_atomics(int *data, int *results) {
	int aux[INDEX_NUM];
	for (int i = 0; i < INDEX_NUM; i++) aux[i] = 0;

	aux[INDEX_MIN] = NUM_MAX;

	for (int i = 0; i < BLOCK_SIZE; i++) {
		aux[INDEX_SUM] += data[blockIdx.x * BLOCK_SIZE + i];
		
		if (data[blockIdx.x * BLOCK_SIZE + i] > aux[INDEX_MAX])
			aux[INDEX_MAX] = data[blockIdx.x * BLOCK_SIZE + i];	

		if (data[blockIdx.x * BLOCK_SIZE + i] < aux[INDEX_MIN])
			aux[INDEX_MIN] = data[blockIdx.x * BLOCK_SIZE + i];
	}

	atomicAdd(results + INDEX_SUM, aux[INDEX_SUM]);
	atomicMax(results + INDEX_MAX, aux[INDEX_MAX]);
	atomicMin(results + INDEX_MIN, aux[INDEX_MIN]);
}

// TODO-3 => ./task_full_atomic
// ITEMS_NUM threads do compute, ITEMS_NUM * 3 atomic calls
// all threads to atomics to compute
__global__ void kernel_full_atomics(int *data, int *results) {
	atomicAdd(results + INDEX_SUM, data[blockIdx.x * blockDim.x + threadIdx.x]);
	atomicMax(results + INDEX_MAX, data[blockIdx.x * blockDim.x + threadIdx.x]);
	atomicMin(results + INDEX_MIN, data[blockIdx.x * blockDim.x + threadIdx.x]);
}

int main(void) {
    int expResults[INDEX_NUM];
    
    int *data = NULL;
    hipMallocManaged(&data, ITEMS_NUM * sizeof(int));
    if (data == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        return 1;
    }
    
    // generate data and expected result
    expResults[INDEX_SUM] = 0;
    expResults[INDEX_MAX] = 0;
    expResults[INDEX_MIN] = NUM_MAX;
    
    for(int i = 0; i < ITEMS_NUM; i++) {
        // each generated number is lower than NUM_MAX as value
        data[i] = rand() % NUM_MAX;
        
        expResults[INDEX_SUM] += data[i];
        
        expResults[INDEX_MAX] = (data[i] > expResults[INDEX_MAX]) ?
            data[i] : expResults[INDEX_MAX];
        
        expResults[INDEX_MIN] = (data[i] < expResults[INDEX_MIN]) ?
            data[i] : expResults[INDEX_MIN];
    }
    
    int *results = NULL;
    hipMallocManaged(&results, INDEX_NUM * sizeof(int));
    if (results == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
    	return 1;
    }
   
    // compute 10 times the results
    for(int i = 0; i < 10; i++) {
        
        // init
        results[INDEX_SUM] = 0;
        results[INDEX_MAX] = 0;
        results[INDEX_MIN] = NUM_MAX;
        
#ifdef NO_ATOMIC
        kernel_no_atomics<<< 1 , 1 >>> (data, results);
        cudaDeviceSynchronize();
#endif

#ifdef PARTIAL_ATOMIC
        kernel_partial_atomics<<< ITEMS_NUM / 256 , 1 >>> (data, results);
        cudaDeviceSynchronize();
#endif

#ifdef FULL_ATOMIC
        kernel_full_atomics<<< ITEMS_NUM / 256 , 256 >>> (data, results);
        cudaDeviceSynchronize();
#endif
    }
    
    cout << "SUM: " << results[INDEX_SUM] << endl;
    if(results[INDEX_SUM] != expResults[INDEX_SUM]) {
        cout << "Failed, SUM should be " << expResults[INDEX_SUM] << endl;
    }
    
    cout << "MAX: " << results[INDEX_MAX] << endl;
    if(results[INDEX_MAX] != expResults[INDEX_MAX]) {
        cout << "Failed, MAX should be " << expResults[INDEX_MAX] << endl;
    }
    
    cout << "MIN: " << results[INDEX_MIN] << endl;
    if(results[INDEX_MIN] != expResults[INDEX_MIN]) {
        cout << "Failed, MIN should be " << expResults[INDEX_MIN] << endl;
    }
    
    hipFree(results);
    return 0;
}
