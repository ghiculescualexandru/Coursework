#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

int main(void) {
    hipSetDevice(0);

    int *host_array_a = 0;
    int *host_array_b = 0;

    int *device_array_a = 0;
    int *device_array_b = 0;
    int *device_array_c = 0;

    // TODO 1: Allocate the host's arrays with the specified number of elements:
    // host_array_a => 32M
    // host_array_b => 32M
	host_array_a = (int *) malloc(BUF_32M * sizeof(int));
	host_array_b = (int *) malloc(BUF_32M * sizeof(int));

    // TODO 2: Allocate the device's arrays with the specified number of elements:
    // device_array_a => 32M
    // device_array_b => 32M
    // device_array_c => 2M
	hipMalloc(&device_array_a, BUF_32M * sizeof(int));
	hipMalloc(&device_array_b, BUF_32M * sizeof(int));
	hipMalloc(&device_array_c, BUF_2M * sizeof(int));

    // Check for allocation errors
    if (host_array_a == 0 || host_array_b == 0 || 
        device_array_a == 0 || device_array_b == 0 || 
        device_array_c == 0) {
        printf("[*] Error!\n");
        return 1;
    }

    for (int i = 0; i < BUF_32M; ++i) {
        host_array_a[i] = i % 32;
        host_array_b[i] = i % 2;
    }

    printf("Before swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 3: Copy from host to device
	hipMemcpy(device_array_a, host_array_a, BUF_32M * sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(device_array_b, host_array_b, BUF_32M * sizeof(int),
		hipMemcpyHostToDevice);
    // TODO 4: Swap the buffers (BUF_2M values each iteration)
    // Hint 1: device_array_c should be used as a temporary buffer
    // Hint 2: hipMemcpy
	for (int i = 0; i != BUF_32M; i += BUF_2M) {
		hipMemcpy(device_array_c, device_array_b + i, BUF_2M * sizeof(int),
			hipMemcpyDeviceToDevice);

		hipMemcpy(device_array_b + i, device_array_a + i, BUF_2M * sizeof(int),
			hipMemcpyDeviceToDevice);

		hipMemcpy(device_array_a + i, device_array_c, BUF_2M * sizeof(int),
			hipMemcpyDeviceToDevice);
	}

    // TODO 5: Copy from device to host
	hipMemcpy(host_array_a, device_array_a, BUF_32M * sizeof(int),
		hipMemcpyDeviceToHost);
	hipMemcpy(host_array_b, device_array_b, BUF_32M * sizeof(int),
		hipMemcpyDeviceToHost);

    printf("\nAfter swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 6: Free the memory
	free(host_array_a);
	free(host_array_b);
	hipFree(device_array_a);
	hipFree(device_array_b);
	hipFree(device_array_c);
    return 0;
}
