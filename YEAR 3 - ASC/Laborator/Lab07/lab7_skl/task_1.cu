#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

#define NMAX (1<<20)

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N) a[i] = a[i] % 2;
}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N) a[i] = blockIdx.x;
}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (i < N) a[i] = threadIdx.x;
}


int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    // ~TODO 1~
    // For each device, show some details in the format below, 
    // then set as active device the first one (assuming there
    // is at least CUDA-capable device). Pay attention to the
    // type of the fields in the hipDeviceProp_t structure.
    //
    // Device number: <i>
    //      Device name: <name>
    //      Total memory: <mem>
    //      Memory Clock Rate (KHz): <mcr>
    //      Memory Bus Width (bits): <mbw>
    // 
    // Hint: look for hipGetDeviceProperties and hipSetDevice in
    // the Cuda Toolkit Documentation. 
    for (int i = 0; i < nDevices; ++i) {
        hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, i);
	
	fprintf(stdout, "Device number: <%d>\n", i);
	fprintf(stdout, "\tDevice name: <%s>\n", prop.name);
	fprintf(stdout, "\tTotal memory: <%zu>\n", prop.totalGlobalMem);
	fprintf(stdout, "\tMemory Clock Rate (KHz): <%d>\n", prop.memoryClockRate);
	fprintf(stdout, "\tMemory Bus Width (bits): <%d>\n", prop.memoryBusWidth);
    }

    // ~TODO 2~
    // With information from example_2.cu, allocate an array with
    // integers (where a[i] = i). Then, modify the three kernels
    // above and execute them using 4 blocks, each with 4 threads.
    // Hint: num_elements = block_size * block_no (see example_2)
    //
    // You can use the fill_array_int(int *a, int n) function (from utils)
    // to fill your array as many times you want.
    int *host_array;
    int *device_array;
    hipError_t rc;

    host_array = (int *) malloc(NMAX * sizeof(int));
    rc = hipMalloc(&device_array, NMAX * sizeof(int));

    if (!host_array || !device_array || rc != hipSuccess)
	exit(-1);

    fill_array_int(host_array, NMAX);
    rc = hipMemcpy(device_array, host_array, NMAX * sizeof(int),
	hipMemcpyHostToDevice);
    if (rc != hipSuccess)
	exit(-1); 
    

    // ~TODO 3~
    // Execute kernel_parity_id kernel and then copy from 
    // the device to the host; call hipDeviceSynchronize()
    // after a kernel execution for safety purposes.
    //
    // Uncomment the line below to check your results
    kernel_parity_id<<<NMAX / 4, 4>>>(device_array, NMAX);
    rc = hipMemcpy(host_array, device_array, NMAX * sizeof(int),
	hipMemcpyDeviceToHost);
    check_task_1(3, host_array);

    // ~TODO 4~
    // Execute kernel_block_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results
    kernel_block_id<<<NMAX / 4, 4>>>(device_array, NMAX);
    rc = hipMemcpy(host_array, device_array, NMAX * sizeof(int),
	hipMemcpyDeviceToHost);
    check_task_1(4, host_array);

    // ~TODO 5~
    // Execute kernel_thread_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results
    kernel_thread_id<<<NMAX / 4, NMAX>>>(device_array, NMAX);
    rc = hipMemcpy(host_array, device_array, NMAX * sizeof(int),
	hipMemcpyDeviceToHost);
    check_task_1(5, host_array);

    // TODO 6: Free the memory
    free(host_array);
    hipFree(device_array);

    return 0;
}
