#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)	c[i] = a[i] + b[i];    
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays
	host_array_a = (float *) malloc(N * sizeof(float));
	host_array_b = (float *) malloc(N * sizeof(float));
	host_array_c = (float *) malloc(N * sizeof(float));
	if (!host_array_a || !host_array_b || !host_array_c)
		exit(-1);

    // TODO 2: Allocate the device's arrays
	hipError_t rc;
	rc = hipMalloc(&device_array_a, N * sizeof(float));
	if (rc != hipSuccess)	exit(-1);
	rc = hipMalloc(&device_array_b, N * sizeof(float));
        if (rc != hipSuccess)  exit(-1);
	rc = hipMalloc(&device_array_c, N * sizeof(float));
        if (rc != hipSuccess)  exit(-1);

    // TODO 3: Check for allocation errors

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.
	fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);

    // TODO 5: Copy the host's arrays to device
	rc = hipMemcpy(device_array_a, host_array_a, N * sizeof(float),
		hipMemcpyHostToDevice);
	if (rc != hipSuccess)	exit(-1);
	rc = hipMemcpy(device_array_b, host_array_b, N * sizeof(float),
                hipMemcpyHostToDevice);
        if (rc != hipSuccess)  exit(-1);

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).
	int num_blocks = N / 256;
	if (N % 256)
		num_blocks++;

	add_arrays<<<num_blocks, 256>>>(device_array_a, device_array_b, device_array_c, N);

    // TODO 7: Copy back the results and then uncomment the checking function
	rc = hipMemcpy(host_array_c, device_array_c, N * sizeof(float),
		hipMemcpyDeviceToHost);
	if (rc != hipSuccess)	exit(-1);
    check_task_2(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory
 	free(host_array_a);
	free(host_array_b);
	free(host_array_c);
	hipFree(device_array_a);
	hipFree(device_array_b);
	hipFree(device_array_c);  
    return 0;
}
